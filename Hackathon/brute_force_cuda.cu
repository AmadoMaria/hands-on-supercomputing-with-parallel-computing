
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <assert.h>

// 97 to 122 use only lowercase letters
// 65 to 90 use only capital letters
// 48 to 57 use only numbers

#define START_CHAR 48
#define END_CHAR 122
#define MAXIMUM_PASSWORD 20

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess)
    {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__device__ long long my_pow(long long x, int y)
{
    long long res = 1;
    if (y == 0)
        return res;
    else
        return x * my_pow(x, y - 1);
}

__global__ void bruteForce(char *pass, long long size)
{
    char force[MAXIMUM_PASSWORD];
    int palavra[MAXIMUM_PASSWORD];
    int pass_b26[MAXIMUM_PASSWORD];

    long long int pass_decimal = 0;
    int base = END_CHAR - START_CHAR + 2;

    for (int i = 0; i < MAXIMUM_PASSWORD; i++)
        force[i] = '\0';

    for (int i = 0; i < size; i++)
        pass_b26[i] = (int)pass[i] - START_CHAR + 1;

    for (int i = size - 1; i > -1; i--)
    {
        pass_decimal += (long long int)pass_b26[i] * my_pow(base, i);
    }

    long long max = my_pow(base, size);

    long long start = threadIdx.x + blockIdx.x * blockDim.x;
    for (long long idx = start; idx < max; idx += gridDim.x * blockDim.x)
    {
        if (idx > pass_decimal)
            return;
        if (idx == pass_decimal)
        {
            int index = 0;
            char s[MAXIMUM_PASSWORD];

            printf("Password in decimal base: %lli\n", idx);
            while ((idx) > 0)
            {
                s[index++] = START_CHAR + idx % base - 1;
                idx /= base;
            }
            s[index] = '\0';
            printf("Found password: %s\n", s);
            return;
        }
    }
}

int main(int argc, char **argv)
{
    char *password;
    time_t t1, t2;
    double dif;

    checkCuda(hipMallocManaged(&password, sizeof(char) * MAXIMUM_PASSWORD));

    hipError_t syncErr, asyncErr;

    strcpy(password, argv[1]);
    int size = strlen(password);

    int deviceId, numberOfSMs;
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    int number_of_blocks = numberOfSMs * 32;
    int threads_per_block = 1024;

    printf("Try to broke the password: %s\n", password);

    time(&t1);
    bruteForce<<<number_of_blocks, threads_per_block>>>(password, size);
    syncErr = hipGetLastError();
    asyncErr = hipDeviceSynchronize();
    time(&t2);

    if (syncErr != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(syncErr));
    if (asyncErr != hipSuccess)
        printf("Error: %s\n", hipGetErrorString(asyncErr));

    dif = difftime(t2, t1);

    printf("\n%1.20f seconds\n", dif);
    hipFree(password);

    return 0;
}