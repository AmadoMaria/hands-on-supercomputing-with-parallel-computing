/****************************************************************************80
 *  Code:
 *   saxpy.c
 *
 *  Purpose:
 *   Implements in C the simple SAXPY stands for Single-Precision.
 *
 *  Modified:
 *   May 08 2022 17:28
 *
 *  Author:
 *    Murilo Boratto  < muriloboratto 'at' fieb.org.br >
 *
 *  HowtoCompile:
 *    gcc saxpy.c -o saxpy
 *
 *  HowtoExecute:
 *    ./saxpy <size>
 *    ./saxpy  10
 *
 *******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void saxpy(int n, float *x, float *y)
{

  int i = threadIdx.x;
  if (i < n)
    y[i] = x[i] + y[i];
}

void printVector(float *vector, int n)
{

  int i;
  for (i = 0; i < n; ++i)
    printf("%1.0f\t", vector[i]);

  printf("\n\n");
}

void generateVector(float *vector, int n)
{
  int i;
  for (i = 0; i < n; ++i)
    vector[i] = i + 1;
}

int main(int argc, char *argv[])
{
  int n = atoi(argv[1]);
  float *x, *y;
  float *xd, *yd;

  hipMalloc((void **)&xd, sizeof(float) * n);
  hipMalloc((void **)&yd, sizeof(float) * n);

  x = (float *)malloc(sizeof(float) * n);
  y = (float *)malloc(sizeof(float) * n);

  generateVector(x, n);
  printVector(x, n);

  generateVector(y, n);
  printVector(y, n);

  hipMemcpy(xd, x, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(yd, x, sizeof(float) * n, hipMemcpyHostToDevice);

  int NUMBER_OF_BLOCKS = 1;
  int NUMBER_OF_THREADS_PER_BLOCKS = n;

  saxpy<<<NUMBER_OF_BLOCKS, NUMBER_OF_THREADS_PER_BLOCKS>>>(n, xd, yd);

  hipMemcpy(y, yd, sizeof(float) * (n), hipMemcpyDeviceToHost);
  printVector(y, n);

  free(x);
  free(y);
  hipFree(xd);
  hipFree(yd);

  return 0;
}
