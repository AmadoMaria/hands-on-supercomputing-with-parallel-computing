/****************************************************************************80
*  Code: 
*   saxpy-cudaMallocManaged.cu
*
*  Purpose:
*   Implements in C using CUDA the simple SAXPY stands for Single-Precision.
*   Using the unified memory (cudaMallocManaged).
*
*  Modified:
*   May 08 2022 17:28 
*
*  Author:
*    Murilo Boratto  < muriloboratto 'at' fieb.org.br >
*
*  HowtoCompile:
*    nvcc saxpy-cudaMallocManaged.cu -o saxpy 
*   
*  HowtoExecute:
*    ./saxpy-cudaMallocManaged <size> 
*    ./saxpy-cudaMallocManaged  10
*
*******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void saxpy(int n,  float *x, float *y){

 int i = threadIdx.x;

 if(i < n)
   y[i] = x[i] + y[i];

}


void printVector(float *vector, int n){

for (int i=0; i < n ; ++i)
 printf("%1.0f\t", vector[i]);

 printf("\n\n");

}


void generateVector(float *vector, int n){

for (int i=0; i < n ; ++i)
 vector[i] = i + 1;

}


int main(int argc, char *argv[]){

  int n = atoi(argv[1]);   
  float *x,*y;

  hipMallocManaged(&x, sizeof(float) * n);
  hipMallocManaged(&y, sizeof(float) * n);
 
  generateVector(x, n);
  printVector(x, n);

  generateVector(y, n);
  printVector(y, n);
  
	saxpy <<< 1, n >>> (n, x, y);
  
  hipDeviceSynchronize();

  printVector(y, n);
 
  hipFree(x);
  hipFree(y);

  return 0;

}

